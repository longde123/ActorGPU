#include "hip/hip_runtime.h"
/*
 Main testing file
 Making sure that our tests run, that things work, etc.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "actor.h"
#include "SchellingActor.hxx"

__global__ void init(Actor* input){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	input[idx] = new SchellingActor();
	__syncthreads();
}

__global__ void sim(Actor* input){
	int idx = threadIdx.x;
	if (idx < 1){
		//input[idx].test = 'a';
		input[idx].react();
	}
	__syncthreads();
}

int main() {
	SchellingActor* schelling_actor_d;
	SchellingActor* schelling_actor_h = new SchellingActor();
	hipMalloc((void**)&schelling_actor_d, sizeof(SchellingActor));
	sim<<<1, 1>>>(schelling_actor_d);
	hipMemcpy(schelling_actor_h, schelling_actor_d, sizeof(SchellingActor), hipMemcpyDeviceToHost);
	printf("schelling_actor_h: %c\n", schelling_actor_h->type());
	hipFree(schelling_actor_d);
}

/*
int main(){
	Actor* wit_actor = new WitActor();
	Actor* mic_actor = new MicActor();

	ActorSystem* system = new ActorSystem();

	system.addActor(mic_actor);
	system.addActor(wit_actor);

	system.simulate();
}*/
